#include "hip/hip_runtime.h"
//#pragma once
#include "management.cuh"
#include "minsw.cuh"
#include"encode.h"
#include <opencv2/photo.hpp>

int h_image_width_ = 0;
int h_image_height_ = 0;
 
bool merge_brightness_flag_ = true;

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


dim3 threadsPerBlock(8, 8);
dim3 blocksPerGrid((d_image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
(d_image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);


// int d_image_width_ = 1920;
// int d_image_height_ = 1200;
// bool load_calib_data_flag_ = false; 

SystemConfigDataStruct cuda_system_config_settings_machine_;
void cuda_set_param_system_config(SystemConfigDataStruct param)
{
	cuda_system_config_settings_machine_ = param;
}

bool cuda_set_projector_version(int version)
{
    switch (version)
    {
    case DF_PROJECTOR_3010:
    {
		int dlp_width = 1280;
		int dlp_height = 720;
		cuda_set_param_dlp_resolution(dlp_width,dlp_height);
 

        return true;
    }
    break;

    case DF_PROJECTOR_4710:
    {
		int dlp_width = 1920;
		int dlp_height = 1080;
 
		cuda_set_param_dlp_resolution(dlp_width,dlp_height);

 
        return true;
    }
    break;

    default:
        break;
    }

	return false;
}

bool cuda_set_camera_resolution(int width,int height)
{
	h_image_width_ = width;
	h_image_height_ = height;
 
	d_image_width_ = width;
	d_image_height_ = height;

	blocksPerGrid.x = (width + threadsPerBlock.x - 1) / threadsPerBlock.x;
	blocksPerGrid.y = (height + threadsPerBlock.y - 1) / threadsPerBlock.y;
 

	return true;
}


//分配basic内存
bool cuda_malloc_basic_memory()
{
    for (int i = 0; i < MAX_PATTERNS_NUMBER; i++)
    {
        hipMalloc((void **)&d_patterns_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }
    for (int i = 0; i < LAST_STEPS_NUM; i++)
    {
        hipMalloc((void **)&d_six_step_patterns_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }
    for (int i = 0; i < LAST_STEPS_NUM; i++)
    {
        hipMalloc((void **)&d_six_step_patterns_convolved_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned char)); 
    }
	for (int i = 0; i < LAST_STEPS_NUM; i++)
    {
        hipMalloc((void **)&d_repetition_02_merge_patterns_convolved_list_[i], d_image_height_ * d_image_width_ * sizeof(unsigned short)); 
    }

    // hipBindTexture(0,texture_patterns_0,d_patterns_list_[0]);
	// hipBindTexture(0,texture_patterns_1,d_patterns_list_[1]);
	// hipBindTexture(0,texture_patterns_2,d_patterns_list_[2]);
	// hipBindTexture(0,texture_patterns_3,d_patterns_list_[3]);
	// hipBindTexture(0,texture_patterns_4,d_patterns_list_[4]);
	// hipBindTexture(0,texture_patterns_5,d_patterns_list_[5]);
	// hipBindTexture(0,texture_patterns_6,d_patterns_list_[6]);
	// hipBindTexture(0,texture_patterns_7,d_patterns_list_[7]);
	// hipBindTexture(0,texture_patterns_8,d_patterns_list_[8]);
	// hipBindTexture(0,texture_patterns_9,d_patterns_list_[9]);
	// hipBindTexture(0,texture_patterns_10,d_patterns_list_[10]);
	// hipBindTexture(0,texture_patterns_11,d_patterns_list_[11]);
	// hipBindTexture(0,texture_patterns_12,d_patterns_list_[12]);
	// hipBindTexture(0,texture_patterns_13,d_patterns_list_[13]);
	// hipBindTexture(0,texture_patterns_14,d_patterns_list_[14]);
	// hipBindTexture(0,texture_patterns_15,d_patterns_list_[15]);
	// hipBindTexture(0,texture_patterns_16,d_patterns_list_[16]);
	// hipBindTexture(0,texture_patterns_17,d_patterns_list_[17]);
	// hipBindTexture(0,texture_patterns_18,d_patterns_list_[18]);

 

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_wrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{
		hipMalloc((void**)&d_unwrap_map_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	} 

	for (int i = 0; i< 2; i++)
	{
		hipMalloc((void**)&d_six_step_pattern_convolution_phase_list_[i], d_image_height_*d_image_width_ * sizeof(float)); 
	} 

	hipMalloc((void**)&d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char));
	hipMalloc((void**)&d_brightness_short_map_, d_image_height_*d_image_width_ * sizeof(unsigned short));  

	hipMalloc((void**)&d_mask_map_, d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	hipMalloc((void**)&d_fisher_mask_, d_image_height_ * d_image_width_ * sizeof(unsigned char));


	hipMalloc((void**)&d_camera_intrinsic_, 3*3 * sizeof(float));
	hipMalloc((void**)&d_project_intrinsic_, 3 * 3 * sizeof(float));

	hipMalloc((void**)&d_camera_distortion_, 1* 5 * sizeof(float));
	hipMalloc((void**)&d_projector_distortion_, 1 * 5 * sizeof(float));

	hipMalloc((void**)&d_rotation_matrix_, 3 * 3 * sizeof(float));
	hipMalloc((void**)&d_translation_matrix_, 1 * 3 * sizeof(float));
 

	hipMalloc((void**)&d_minsw8_table_, 256* sizeof(unsigned char));

	unsigned char array[256] = { 0, 1, 105, 2, 155, 154, 156, 3, 19, 172, 106, 107, 70, 69, 157, 4, 169,
		66, 192, 193, 220, 67, 91, 90, 170, 171, 211, 108, 221, 68, 6, 5, 255, 152, 126, 23, 50, 
		153, 177, 176, 20, 173, 21, 22, 71, 174, 72, 175, 190, 87, 191, 88, 241, 240, 242, 89, 85,
		86, 212, 109, 136, 239, 7, 110, 233, 130, 128, 129, 28, 131, 27, 26, 234, 235, 147, 44, 29,
		132, 198, 197, 64, 65, 41, 194, 219, 218, 92, 195, 83, 236, 42, 43, 134, 133, 93, 196, 254,
		151, 127, 24, 49, 48, 178, 25, 149, 150, 148, 45, 200, 47, 199, 46, 63, 216, 62, 215, 114, 
		217, 113, 112, 84, 237, 213, 214, 135, 238, 8, 111, 103, 206, 104, 207, 52, 205, 53, 54, 
		18, 121, 209, 208, 223, 120, 158, 55, 168, 15, 39, 142, 117, 118, 244, 141, 17, 16, 210, 57,
		222, 119, 159, 56, 102, 101, 125, 228, 51, 204, 74, 75, 123, 122, 124, 227, 224, 225, 73, 
		226, 189, 36, 38, 37, 138, 139, 243, 140, 188, 35, 59, 58, 137, 34, 160, 161, 232, 79, 231,
		78, 181, 182, 180, 77, 81, 80, 146, 249, 30, 183, 95, 248, 167, 14, 40, 143, 116, 13, 245,
		246, 82, 185, 145, 144, 31, 184, 94, 247, 253, 100, 230, 229, 202, 203, 179, 76, 252, 99, 
		251, 250, 201, 98, 96, 97, 166, 165, 61, 164, 115, 12, 10, 11, 187, 186, 60, 163, 32, 33, 9, 162 };

		
	CHECK(hipMemcpy(d_minsw8_table_, array, 256 * sizeof(unsigned char), hipMemcpyHostToDevice));

	hipMalloc((void**)&d_fisher_confidence_map, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_convolution_kernal_map, SIZE_OF_CONVOLUTION_KERNAL*SIZE_OF_CONVOLUTION_KERNAL * sizeof(float));
	hipMalloc((void**)&d_point_cloud_map_, 3*d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_temp_, d_image_height_*d_image_width_ * sizeof(float));
	hipMalloc((void**)&d_triangulation_error_map_, d_image_height_*d_image_width_ * sizeof(float));

	
	hipMalloc((void**)&d_global_light_map_, d_image_height_*d_image_width_ * sizeof(char));
	hipMalloc((void**)&d_direct_light_map_, d_image_height_*d_image_width_ * sizeof(char));
	hipMalloc((void**)&d_uncertain_map_, d_image_height_*d_image_width_ * sizeof(char));
 
 	hipMalloc((void**)&d_single_pattern_mapping_, 4000*2000 * sizeof(float)); 
	hipMalloc((void**)&d_single_pattern_minimapping_, 128*128 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

	hipMalloc((void**)&d_undistort_map_x_, d_image_height_*d_image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_undistort_map_y_, d_image_height_*d_image_width_ * sizeof(float)); 

    LOG(INFO)<<"d_image_width_: "<<d_image_width_;
    LOG(INFO)<<"d_image_height_: "<<d_image_height_;
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_basic_memory()
{

	for (int i = 0; i< MAX_PATTERNS_NUMBER; i++)
	{  
		hipFree(d_patterns_list_[i]); 
	}
	for (int i = 0; i< LAST_STEPS_NUM; i++)
	{  
		hipFree(d_six_step_patterns_list_[i]); 
	}
	for (int i = 0; i< LAST_STEPS_NUM; i++)
	{  
		hipFree(d_six_step_patterns_convolved_list_[i]); 
	}
	for (int i = 0; i< LAST_STEPS_NUM; i++)
	{  
		hipFree(d_repetition_02_merge_patterns_convolved_list_[i]); 
	}
	// hipUnbindTexture(texture_patterns_0);
	// hipUnbindTexture(texture_patterns_1);
	// hipUnbindTexture(texture_patterns_2);
	// hipUnbindTexture(texture_patterns_3);
	// hipUnbindTexture(texture_patterns_4);
	// hipUnbindTexture(texture_patterns_5);
	// hipUnbindTexture(texture_patterns_6);
	// hipUnbindTexture(texture_patterns_7);
	// hipUnbindTexture(texture_patterns_8);
	// hipUnbindTexture(texture_patterns_9);
	// hipUnbindTexture(texture_patterns_10);
	// hipUnbindTexture(texture_patterns_11);
	// hipUnbindTexture(texture_patterns_12);
	// hipUnbindTexture(texture_patterns_13);
	// hipUnbindTexture(texture_patterns_14);
	// hipUnbindTexture(texture_patterns_15);
	// hipUnbindTexture(texture_patterns_16);
	// hipUnbindTexture(texture_patterns_17);
	// hipUnbindTexture(texture_patterns_18);

	for (int i = 0; i< MAX_WRAP_NUMBER; i++)
	{  
		hipFree(d_wrap_map_list_[i]);
		hipFree(d_confidence_map_list_[i]); 
	}

	for (int i = 0; i< MAX_UNWRAP_NUMBER; i++)
	{ 
		hipFree(d_unwrap_map_list_[i]); 
	}

	for (int i = 0; i< 2; i++)
	{ 
		hipFree(d_six_step_pattern_convolution_phase_list_[i]); 
	}

	hipFree(d_fisher_confidence_map);
	hipFree(d_convolution_kernal_map);
	hipFree(d_fisher_mask_);
    hipFree(d_mask_map_);
    hipFree(d_brightness_map_);
    hipFree(d_brightness_short_map_);

    hipFree(d_point_cloud_map_);
    hipFree(d_depth_map_);
	hipFree(d_depth_map_temp_);
    hipFree(d_triangulation_error_map_);

    hipFree(d_global_light_map_);
    hipFree(d_direct_light_map_);
    hipFree(d_uncertain_map_);
	


    hipFree(d_camera_intrinsic_);
	hipFree(d_project_intrinsic_); 
	hipFree(d_camera_distortion_);
	hipFree(d_projector_distortion_); 
	hipFree(d_rotation_matrix_);
	hipFree(d_translation_matrix_);

    hipFree(d_minsw8_table_);
	
 
    hipFree(d_single_pattern_mapping_);
    hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);

    hipFree(d_undistort_map_x_);
    hipFree(d_undistort_map_y_);
 

	return true;
}

 //分配hdr内存
bool cuda_malloc_hdr_memory()
{
	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{
		hipMalloc((void**)&d_hdr_depth_map_list_[i], d_image_height_*d_image_width_ * sizeof(float));
		hipMalloc((void**)&d_hdr_brightness_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char));  
		hipMalloc((void**)&d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float)); 
	}
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_hdr_memory()
{
    for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{ 
		hipFree(d_hdr_depth_map_list_[i]);
		hipFree(d_hdr_brightness_list_[i]); 
		hipFree(d_hdr_bright_pixel_sum_list_[i]);
	}
	
	return true;
}

//分配repetition内存
bool cuda_malloc_repetition_memory()
{
	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipMalloc((void**)&d_repetition_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned char)); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipMalloc((void**)&d_repetition_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}
 
 	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipMalloc((void**)&d_repetition_02_merge_patterns_list_[i], d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	}

	hipMalloc((void**)&d_merge_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned short)); 
	
	hipDeviceSynchronize();
	return true;
}

bool cuda_free_repetition_memory()
{

	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipFree(d_repetition_patterns_list_[i]); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipFree(d_repetition_merge_patterns_list_[i]);  
	}

	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipFree(d_repetition_02_merge_patterns_list_[i]);  
	}

	hipFree(d_merge_brightness_map_);  
	
	return true;
}


/********************************************************************************************/
//copy 
void cuda_copy_calib_data(float* camera_intrinsic, float* project_intrinsic, float* camera_distortion,
	float* projector_distortion, float* rotation_matrix, float* translation_matrix)
{
  
	CHECK(hipMemcpy(d_camera_intrinsic_, camera_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_project_intrinsic_, project_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_camera_distortion_, camera_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_projector_distortion_, projector_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_rotation_matrix_, rotation_matrix, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_translation_matrix_, translation_matrix, 1* 3 * sizeof(float), hipMemcpyHostToDevice));

	load_calib_data_flag_ = 1;

 
}

void cuda_copy_talbe_to_memory(float* mapping,float* mini_mapping,float* rotate_x,float* rotate_y,float* r_1,float base_line)
{
   
	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3*3 * sizeof(float), hipMemcpyHostToDevice)); 
	CHECK(hipMemcpyAsync(d_single_pattern_minimapping_, mini_mapping, 128 * 128 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_single_pattern_mapping_, mapping, 4000*2000 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	
    d_baseline_ = base_line;  
 

	LOG(INFO)<<"d_baseline_: "<<d_baseline_;
	hipDeviceSynchronize();
}


void coud_copy_undistort_table_to_memory(float* undistort_x_map,float* undistort_y_map)
{
	CHECK(hipMemcpyAsync(d_undistort_map_x_, undistort_x_map, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_undistort_map_y_, undistort_y_map, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyHostToDevice));
}


bool cuda_copy_pattern_to_memory(unsigned char* pattern_ptr,int serial_flag)
{
	if(serial_flag> MAX_PATTERNS_NUMBER)
	{
		return false;
	}

	cv::Mat smooth_mat(d_image_height_, d_image_width_, CV_8UC1, pattern_ptr);
	if (serial_flag < 12)
	{
		LOG(INFO) << "Start GaussianBlur:";
		cv::GaussianBlur(smooth_mat, smooth_mat, cv::Size(5, 5), 1, 1);

		LOG(INFO) << "finished GaussianBlur!";
	}

	CHECK(hipMemcpyAsync(d_patterns_list_[serial_flag], smooth_mat.data, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

void cuda_copy_pointcloud_from_memory(float* pointcloud)
{ 
	CHECK(hipMemcpy(pointcloud, d_point_cloud_map_, 3 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
}

void cuda_copy_depth_from_memory(float* depth)
{
	CHECK(hipMemcpy(depth, d_depth_map_, d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
} 

void cuda_copy_brightness_from_memory(unsigned char* brightness)
{
	CHECK(hipMemcpy(brightness, d_brightness_map_, d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost)); 
}

void cuda_copy_convolution_kernal_to_memory(float* convolution_kernal, int kernal_diameter)
{
	CHECK(hipMemcpyAsync(d_convolution_kernal_map, convolution_kernal, kernal_diameter*kernal_diameter* sizeof(float), hipMemcpyHostToDevice)); 
}


void cuda_copy_brightness_16_to_memory(unsigned short* brightness)
{
	CHECK(hipMemcpyAsync(d_brightness_short_map_, brightness, d_image_height_*d_image_width_* sizeof(unsigned short), hipMemcpyHostToDevice)); 

}

void cuda_copy_brightness_to_memory(unsigned char* brightness)
{ 
	CHECK(hipMemcpyAsync(d_brightness_map_, brightness, d_image_height_*d_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice)); 
}


void cuda_clear_reconstruct_cache()
{
	
	CHECK(hipMemset(d_brightness_map_,0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_depth_map_,0, d_image_height_*d_image_width_ * sizeof(float))); 
	CHECK(hipMemset(d_point_cloud_map_,0,3* d_image_height_*d_image_width_ * sizeof(float))); 
	CHECK(hipMemset(d_mask_map_,0,d_image_height_*d_image_width_ * sizeof(unsigned char))); 
    CHECK(hipMemset(d_fisher_confidence_map,0,d_image_height_*d_image_width_ * sizeof(float))); 

	CHECK(hipMemset(d_hdr_brightness_list_[0],0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_hdr_brightness_list_[1],0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_hdr_brightness_list_[2],0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_hdr_brightness_list_[3],0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_hdr_brightness_list_[4],0, d_image_height_*d_image_width_ * sizeof(char))); 
	CHECK(hipMemset(d_hdr_brightness_list_[5],0, d_image_height_*d_image_width_ * sizeof(char)));

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{
		CHECK(hipMemset(d_direct_light_map_, 0, d_image_height_ * d_image_width_ * sizeof(char)));
		CHECK(hipMemset(d_global_light_map_, 0, d_image_height_ * d_image_width_ * sizeof(char)));
		CHECK(hipMemset(d_uncertain_map_, 0, d_image_height_ * d_image_width_ * sizeof(char)));
	}
}


/********************************************************************************************/


bool cuda_compute_phase_shift(int serial_flag)
{
	 
	switch(serial_flag)
	{
		case 0:
		{ 
        	LOG(INFO)<<"kernel_four_step_phase_shift:"<<d_image_width_;
			int i= 0;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);

				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 1:
		{

			int i= 4;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
			
		}
		break;
		case 2:
		{ 
			int i= 8;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				
				// kernel_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 3:
		{ 
			int i= 12; 
			kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3],d_patterns_list_[i + 4],d_patterns_list_[i + 5] ,d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
			kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_, d_image_height_, d_patterns_list_[i + 0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_patterns_list_[i + 4], d_patterns_list_[i + 5], d_depth_map_, d_confidence_map_list_[serial_flag]);

			if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
			{

				kernel_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[i + 0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
																				 d_patterns_list_[i + 3], d_patterns_list_[i + 4], d_patterns_list_[i + 5],
																				 cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
			
			
			
			}

				// cuda_six_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> > (d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
				// hipDeviceSynchronize();

				// cv::Mat phase(1200, 1920, CV_32F, cv::Scalar(0));
				// CHECK(hipMemcpy(phase.data, d_wrap_map_list_[serial_flag], 1 * image_height_ * image_width_ * sizeof(float), hipMemcpyDeviceToHost));
				// cv::imwrite("phase1.tiff",phase);
		}
		break;
		case 4:
		{
			int i= 18;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 5:
		{
			int i= 22;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
		case 6:
		{
			int i= 26;
			kernel_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0], d_patterns_list_[i + 1], d_patterns_list_[i + 2],
				d_patterns_list_[i + 3], d_wrap_map_list_[serial_flag], d_confidence_map_list_[serial_flag]);
		}
		break;
  
		default :
			break;
	}

	
	
	return true;
}

bool cuda_compute_convolved_image_phase_shift(int serial_flag)
{
	switch (serial_flag)
	{
	case 0:
	{
		kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,
		d_six_step_patterns_convolved_list_[0], d_six_step_patterns_convolved_list_[1], 
		d_six_step_patterns_convolved_list_[2], d_six_step_patterns_convolved_list_[3],
		d_six_step_patterns_convolved_list_[4],d_six_step_patterns_convolved_list_[5],
		d_six_step_pattern_convolution_phase_list_[1], d_confidence_map_list_[3]);
	}
		break;

	case 1:
	{
		kernel_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_convolved_list_[0],
		 d_repetition_02_merge_patterns_convolved_list_[1], d_repetition_02_merge_patterns_convolved_list_[2],
		  d_repetition_02_merge_patterns_convolved_list_[3],d_repetition_02_merge_patterns_convolved_list_[4],
		  d_repetition_02_merge_patterns_convolved_list_[5], 1, d_image_height_, d_image_width_, 
		  d_six_step_pattern_convolution_phase_list_[1], d_confidence_map_list_[3]);
	}
		break;
	
	default:
		break;
	}


	return true;
}

bool cuda_rectify_six_step_pattern_phase(int mode, int kernal_diameter)
{	
	switch (mode)
	{
		case 0:
		{
			hipDeviceSynchronize();
			LOG(INFO)<<"start six_step blur mode 0";
			for (int i = 0; i < LAST_STEPS_NUM; i += 1)
			{
				kernal_convolution_2D<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_, d_patterns_list_[12 + i],
				 d_six_step_patterns_convolved_list_[i], d_convolution_kernal_map, kernal_diameter);
			}
			hipDeviceSynchronize();
			LOG(INFO)<<"end six_step blur";

			cuda_compute_convolved_image_phase_shift(0);

			// 计算相位并且补偿
			kernel_six_step_phase_rectify<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[3], 
			d_six_step_pattern_convolution_phase_list_[1], d_wrap_map_list_[3]);
		}
		break;

		case 1:
		{
			hipDeviceSynchronize();
			LOG(INFO)<<"start six_step blur mode 1";
			for (int i = 0; i < LAST_STEPS_NUM; i += 1)
			{
				kernal_convolution_2D_short<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,
				 d_repetition_02_merge_patterns_list_[12 + i], d_repetition_02_merge_patterns_convolved_list_[i],
				  d_convolution_kernal_map, kernal_diameter);
			}
			hipDeviceSynchronize();
			LOG(INFO)<<"end six_step blur";

			cuda_compute_convolved_image_phase_shift(1);

			// 计算相位并且补偿
			kernel_six_step_phase_rectify<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[3],
			 d_six_step_pattern_convolution_phase_list_[1], d_wrap_map_list_[3]);
		}
		break;
		case 2:
		{
			//minsw相位校正
			hipDeviceSynchronize();
			LOG(INFO)<<"start six_step blur mode 0";
			for (int i = 0; i < LAST_STEPS_NUM; i += 1)
			{
				kernal_convolution_2D<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,
				 d_patterns_list_[2 + i], d_six_step_patterns_convolved_list_[i], d_convolution_kernal_map, kernal_diameter);
			}
			hipDeviceSynchronize();
			LOG(INFO)<<"end six_step blur";

			cuda_compute_convolved_image_phase_shift(0);

			// 计算相位并且补偿
			kernel_six_step_phase_rectify<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[3],
			 d_six_step_pattern_convolution_phase_list_[1], d_wrap_map_list_[3]);
		}
		break; 
		case 3:
		{
			//minsw相位校正repetition
			hipDeviceSynchronize();
			LOG(INFO)<<"start six_step blur mode 1";
			for (int i = 0; i < LAST_STEPS_NUM; i += 1)
			{
				kernal_convolution_2D_short<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,
				 d_repetition_02_merge_patterns_list_[2 + i], d_repetition_02_merge_patterns_convolved_list_[i],
				  d_convolution_kernal_map, kernal_diameter);
			}
			hipDeviceSynchronize();
			LOG(INFO)<<"end six_step blur";

			cuda_compute_convolved_image_phase_shift(1);

			// 计算相位并且补偿
			kernel_six_step_phase_rectify<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,
			d_wrap_map_list_[3], d_six_step_pattern_convolution_phase_list_[1], d_wrap_map_list_[3]);
		}
		break;
		default:
			break;
	}

	
}

bool cuda_normalize_phase(int serial_flag)
{
    switch(serial_flag)
	{ 
        case 0:
		{   
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], (float)128.0, d_unwrap_map_list_[0]);  
		}
		break; 
		case 1:
		{   
  
            kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)18., d_unwrap_map_list_[1]); 
		}
		break;

		case 2:
		{ 
			kernel_normalize_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], (float)72., d_unwrap_map_list_[1]); 
		}
		break;

		default :
			break;
	}


	return true;
}

bool cuda_unwrap_phase_shift(int serial_flag)
{

	switch(serial_flag)
	{ 
		case 1:
		{  
            kernel_unwrap_variable_phase<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[0], d_wrap_map_list_[1], 8.0, CV_PI, d_unwrap_map_list_[0]);
  
		}
		break;

		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[2], 4.0,CV_PI, d_unwrap_map_list_[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[3], 4.0,1.5, d_unwrap_map_list_[0]); 
 
		}
		break;
		case 4:
		{
 
		}
		break;
		case 5:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[4], d_wrap_map_list_[5], 8.0,CV_PI, d_unwrap_map_list_[1]);
		}
		break;
		case 6:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[6], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
			LOG(INFO)<<"unwrap 6:  ";

		}
		break;
		case 7:
		{
			kernel_unwrap_variable_phase << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[1], d_wrap_map_list_[7], 4.0,CV_PI, d_unwrap_map_list_[1]);
 
		 	LOG(INFO)<<"unwrap 7:  ";

		}
		break;
 

		default :
			break;
	}


	return true;
}

bool cuda_unwrap_phase_shift_base_fisher_confidence(int serial_flag)
{

	switch(serial_flag)
	{ 
		case 1:
		{  
            kernel_unwrap_variable_phase_base_confidence<< <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_wrap_map_list_[0], d_wrap_map_list_[1], 8.0, CV_PI, FISHER_RATE_1, d_fisher_confidence_map, d_unwrap_map_list_[0]);
  
		}
		break;

		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase_base_confidence << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[2], 4.0,CV_PI, FISHER_RATE_2, d_fisher_confidence_map, d_unwrap_map_list_[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_unwrap_variable_phase), hipFuncCachePreferL1) );
			kernel_unwrap_variable_phase_base_confidence << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_unwrap_map_list_[0], d_wrap_map_list_[3], 4.0,1.5, FISHER_RATE_3, d_fisher_confidence_map, d_unwrap_map_list_[0]); 
 
		}
		break;
		default :
			break;
	}


	return true;
}

/********************************************************************************************************************************************/

bool cuda_generate_pointcloud_base_minitable()
{
		if(1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_reflect_filter)
	{ 
		LOG(INFO)<<"filter_reflect_noise start:"; 
		cuda_filter_reflect_noise(d_unwrap_map_list_[0]); 

		hipDeviceSynchronize();
		LOG(INFO)<<"filter_reflect_noise end";
	}

	kernel_reconstruct_pointcloud_base_minitable<< <blocksPerGrid, threadsPerBlock>> > (d_image_width_,d_image_height_,d_xL_rotate_x_,d_xL_rotate_y_,d_single_pattern_minimapping_,d_R_1_,d_baseline_,
	d_confidence_map_list_[3],d_unwrap_map_list_[0],d_point_cloud_map_,d_depth_map_);

 
}


bool cuda_merge_brigntness(int hdr_num, unsigned char* brightness)
{
	if(!merge_brightness_flag_)
	{ 
		return false;
	} 

	std::vector<cv::Mat> brightness_list;
	cv::Mat image_b(h_image_height_, h_image_width_, CV_8U, cv::Scalar(0));

	hipDeviceSynchronize();
	for (int i = 0; i < hdr_num; i++)
	{

		CHECK(hipMemcpy(image_b.data, d_hdr_brightness_list_[i], 1 * h_image_height_ * h_image_width_ * sizeof(uchar), hipMemcpyDeviceToHost));
		brightness_list.push_back(image_b.clone());
	}

	LOG(INFO) << "process: "<<hdr_num;
	cv::Mat exposureFusion;
	cv::Ptr<cv::MergeMertens> mergeMertens = cv::createMergeMertens();
	mergeMertens->process(brightness_list, exposureFusion);
  
	for (int r = 0; r < h_image_height_; r++)
	{ 
		float *ptr_fusion = exposureFusion.ptr<float>(r);

		for (int c = 0; c < h_image_width_; c++)
		{
			if (ptr_fusion[c] > 1)
			{ 
				brightness[r*h_image_width_+c] = 255;
			}
			else
			{ 
				brightness[r*h_image_width_+c] = 255 * ptr_fusion[c];
			}
		}
	}

	LOG(INFO) << "get exposureFusion!"; 
}

bool cuda_generate_pointcloud_base_table()
{
	// cv::Mat phase(d_image_height_,d_image_width_,CV_32FC1,cv::Scalar(0));
	// CHECK(hipMemcpy(phase.data, d_unwrap_map_list_[0], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// cv::imwrite("phase.tiff", phase);
	
	// if(1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	// { 
	// 	LOG(INFO)<<"filter_reflect_noise start:"; 
	// 	cuda_filter_reflect_noise(d_unwrap_map_list_[0]); 
	
	// 	// hipDeviceSynchronize();
	// 	LOG(INFO)<<"filter_reflect_noise end";
	// }



	kernel_reconstruct_pointcloud_base_table<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_xL_rotate_x_, d_xL_rotate_y_, d_single_pattern_mapping_, d_R_1_, d_baseline_,
																				 d_confidence_map_list_[3], d_unwrap_map_list_[0], d_point_cloud_map_, d_depth_map_);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{
		kernel_remove_mask_result<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_uncertain_map_,
		cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_threshold,d_depth_map_,d_point_cloud_map_);

		// hipDeviceSynchronize();

		// cv::Mat uncertain_map(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
		// CHECK(hipMemcpy(uncertain_map.data, d_uncertain_map_, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
		// cv::imwrite("uncertain_map.bmp", uncertain_map);
	}

	// cv::Mat depth(d_image_height_,d_image_width_,CV_32FC1,cv::Scalar(0));
	// CHECK(hipMemcpy(depth.data, d_depth_map_, 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// cv::imwrite("depth.tiff", depth);
	return true;
}

/********************************************************************************************************************************************/


bool cuda_copy_result_to_hdr_color(int serial_flag,int brigntness_serial,cv::Mat brightness)
{
	CHECK(hipMemcpyAsync(d_hdr_brightness_list_[serial_flag], brightness.data, 1 * d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));


	if(!load_calib_data_flag_)
	{
		return false;
	}
 
	// cv::imwrite("brightness.bmp",brightness);

	CHECK(hipMemcpyAsync(d_hdr_depth_map_list_[serial_flag], d_depth_map_, 1 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToDevice)); 

	float val  = 0;
	CHECK(hipMemcpyAsync(d_hdr_bright_pixel_sum_list_[serial_flag], &val, sizeof(float), hipMemcpyHostToDevice)); 
 	cuda_count_sum_pixel << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_list_[serial_flag],d_image_height_,d_image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);
 
	LOG(INFO)<<"cuda_copy_result_to_hdr color: "<<serial_flag;
	return true;
}

// bool cuda_copy_result_to_hdr_16(int serial_flag,int brigntness_serial)
// {
	 
// 	// CHECK(hipMemcpyAsync(d_hdr_brightness_short_list_[serial_flag], d_brightness_short_map_, 1 * d_image_height_*d_image_width_ * sizeof(unsigned short), hipMemcpyDeviceToDevice));
// 	CHECK(hipMemcpyAsync(d_hdr_brightness_list_[serial_flag], d_brightness_map_, 1 * d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToDevice));


// 	if(!load_calib_data_flag_)
// 	{
// 		return false;
// 	}
 

// 	CHECK(hipMemcpyAsync(d_hdr_depth_map_list_[serial_flag], d_depth_map_, 1 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToDevice)); 

// 	float val  = 0;
// 	CHECK(hipMemcpyAsync(d_hdr_bright_pixel_sum_list_[serial_flag], &val, sizeof(float), hipMemcpyHostToDevice)); 
//  	// cuda_count_sum_pixel_16 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_short_list_[serial_flag],d_image_height_,d_image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);
 
//   	cuda_count_sum_pixel << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_list_[serial_flag],d_image_height_,d_image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);
 
// 	LOG(INFO)<<"cuda_copy_result_to_hdr: "<<serial_flag;
// 	return true;

// }

bool cuda_copy_result_to_hdr(int serial_flag,int brigntness_serial)
{
	CHECK(hipMemcpyAsync(d_hdr_brightness_list_[serial_flag], d_brightness_map_, 1 * d_image_height_*d_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToDevice));


	if(!load_calib_data_flag_)
	{
		return false;
	}
 

	CHECK(hipMemcpyAsync(d_hdr_depth_map_list_[serial_flag], d_depth_map_, 1 * d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToDevice)); 

	float val  = 0;
	CHECK(hipMemcpyAsync(d_hdr_bright_pixel_sum_list_[serial_flag], &val, sizeof(float), hipMemcpyHostToDevice)); 
 	cuda_count_sum_pixel << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_list_[serial_flag],d_image_height_,d_image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);

	LOG(INFO)<<"cuda_copy_result_to_hdr: "<<serial_flag;
	return true;
}

bool cuda_merge_hdr_data_16(int hdr_num,float* depth_map, unsigned char* brightness)
{
 

		LOG(INFO) << "sum pixels ";
		float sum_pixels_list[6];

		for (int i = 0; i < hdr_num; i++)
		{
			CHECK(hipMemcpy(&sum_pixels_list[i], d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float), hipMemcpyDeviceToHost));
		}

		std::vector<float> param_list;
		std::vector<int> id;
		std::vector<bool> flag_list;

		for (int i = 0; i < hdr_num; i++)
		{
			param_list.push_back(sum_pixels_list[i]);
			id.push_back(0);
			flag_list.push_back(true);
		}
		std::sort(param_list.begin(), param_list.end(), std::greater<float>());

		for (int i = 0; i < hdr_num; i++)
		{

			for (int j = 0; j < hdr_num; j++)
			{
				if (param_list[i] == sum_pixels_list[j])
				{
					if (flag_list[j])
					{
						id[i] = j;
						flag_list[j] = false;
						break;
					}
				}
			}
		}

		for (int i = 0; i < hdr_num; i++)
		{
			LOG(INFO) << "sum pixels " << i << ": " << sum_pixels_list[i] << " _ " << id[i];
		}

		switch (hdr_num)
		{
		case 1:
		{

			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_list_[0], 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[0], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;
		case 2:
		{
			cuda_merge_hdr_2<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_depth_map_list_[id[0]], d_hdr_depth_map_list_[id[1]], d_hdr_brightness_list_[id[0]],
																 d_hdr_brightness_list_[id[1]], h_image_height_, h_image_width_, d_depth_map_, d_brightness_map_);

			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;
		case 3:
		{
			cuda_merge_hdr_3<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_depth_map_list_[id[0]], d_hdr_depth_map_list_[id[1]], d_hdr_depth_map_list_[id[2]], d_hdr_brightness_list_[id[0]],
																 d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], h_image_height_, h_image_width_, d_depth_map_, d_brightness_map_);

			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;
		case 4:
		{
			cuda_merge_hdr_4<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_depth_map_list_[id[0]], d_hdr_depth_map_list_[id[1]], d_hdr_depth_map_list_[id[2]], d_hdr_depth_map_list_[id[3]],
																 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]],
																 h_image_height_, h_image_width_, d_depth_map_, d_brightness_map_);

			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;
		case 5:
		{
			cuda_merge_hdr_5<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_depth_map_list_[id[0]], d_hdr_depth_map_list_[id[1]], d_hdr_depth_map_list_[id[2]],
																 d_hdr_depth_map_list_[id[3]], d_hdr_depth_map_list_[id[4]],
																 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]],
																 h_image_height_, h_image_width_, d_depth_map_, d_brightness_map_);

			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;
		case 6:
		{
			cuda_merge_hdr_6<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_depth_map_list_[id[0]], d_hdr_depth_map_list_[id[1]], d_hdr_depth_map_list_[id[2]],
																 d_hdr_depth_map_list_[id[3]], d_hdr_depth_map_list_[id[4]], d_hdr_depth_map_list_[id[5]],
																 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]],
																 d_hdr_brightness_list_[id[5]],
																 h_image_height_, h_image_width_, d_depth_map_, d_brightness_map_);

			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		}
		break;

		default:
			return false;
		}

		// kernel_merge_brigntness_map<<<blocksPerGrid, threadsPerBlock>>>(d_hdr_brightness_short_list_[hdr_num - 1], 16,
		// 																h_image_height_, h_image_width_, d_brightness_map_);


		CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[hdr_num - 1], 1*h_image_height_ * h_image_width_  * sizeof(unsigned char), hipMemcpyDeviceToHost));

		// CHECK(hipMemcpy(brightness, d_brightness_map_, 1 * h_image_height_ * h_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		LOG(INFO) << "DHR Finished!";

		// cv::Mat depth(d_image_height_,d_image_width_,CV_32FC1,cv::Scalar(0));
		// CHECK(hipMemcpy(depth.data, d_depth_map_, 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
		// cv::imwrite("depth.tiff", depth);

		// cv::Mat brightness_mat(d_image_height_,d_image_width_,CV_8U,brightness);
		// // CHECK(hipMemcpy(brightness_mat.data, brightness, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
		// cv::imwrite("brightness.tiff", brightness_mat);

		return true;
	
}

bool cuda_merge_hdr_data(int hdr_num,float* depth_map, unsigned char* brightness)
{
	
	LOG(INFO)<<"sum pixels ";
	float sum_pixels_list[6];  

    for(int i= 0;i<hdr_num;i++)
    { 
		CHECK(hipMemcpy(&sum_pixels_list[i], d_hdr_bright_pixel_sum_list_[i], 1* sizeof(float), hipMemcpyDeviceToHost));
    }
 
 
	std::vector<float> param_list;
	std::vector<int> id; 
	std::vector<bool> flag_list;

	for (int i = 0; i < hdr_num; i++)
	{ 
        param_list.push_back(sum_pixels_list[i]);
		id.push_back(0);
		flag_list.push_back(true);
    } 
   	std::sort(param_list.begin(),param_list.end(),std::greater<float>());
 
 
	for (int i = 0; i < hdr_num; i++)
	{ 
		
		for(int j= 0;j< hdr_num;j++)
		{
			if(param_list[i] == sum_pixels_list[j])
			{
				if(flag_list[j])
				{ 
					id[i] = j;
					flag_list[j] = false; 
					break;
				}
			}
		}
		 
    } 

 
	for (int i = 0; i < hdr_num; i++)
	{ 
        LOG(INFO)<<"sum pixels "<<i<<": "<<sum_pixels_list[i]<<" _ "<<id[i];
    }
 

	switch(hdr_num)
	{
		case 1:
		{

			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_list_[0], 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[0], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		} 
		break;
		case 2:
		{
			cuda_merge_hdr_2 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);

				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 3:
		{
			cuda_merge_hdr_3 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 4:
		{
			cuda_merge_hdr_4 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],d_hdr_depth_map_list_[id[3]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 5:
		{
			cuda_merge_hdr_5 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 6:
		{
			cuda_merge_hdr_6 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],d_hdr_depth_map_list_[id[5]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				 d_hdr_brightness_list_[id[5]], 
				h_image_height_, h_image_width_, d_depth_map_,d_brightness_map_);
				
			CHECK(hipMemcpy(depth_map, d_depth_map_, 1 * h_image_height_*h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;

		default:
		 		return false;

	}

 	// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[id[0]], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
 	CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[hdr_num-1], 1*h_image_height_*h_image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
	LOG(INFO)<<"DHR Finished!";

	return true;
}

bool cuda_merge_hdr_data_cpu(int hdr_num,int hight,int width, std::vector<cv::Mat> hdr_brightness_list, std::vector<cv::Mat> hdr_depth_list, std::vector<float> hdr_bright_pixel_sum, cv::Mat& depth, cv::Mat& bright)
{

	LOG(INFO) << "sum pixels ";
	float sum_pixels_list[6];

	for (int i = 0; i < hdr_num; i++)
	{
		sum_pixels_list[i]= hdr_bright_pixel_sum[i];
	}

	std::vector<float> param_list;
	std::vector<int> id;
	std::vector<bool> flag_list;

	for (int i = 0; i < hdr_num; i++)
	{
		param_list.push_back(sum_pixels_list[i]);
		id.push_back(0);
		flag_list.push_back(true);
	}
	std::sort(param_list.begin(), param_list.end(), std::greater<float>());


	for (int i = 0; i < hdr_num; i++)
	{

		for (int j = 0; j < hdr_num; j++)
		{
			if (param_list[i] == sum_pixels_list[j])
			{
				if (flag_list[j])
				{
					id[i] = j;
					flag_list[j] = false;
					break;
				}
			}
		}

	}

	for (int i = 0; i < hdr_num; i++)
	{
		LOG(INFO) << "sum pixels " << i << ": " << sum_pixels_list[i] << " _ " << id[i];
	}

	cv::Mat depthptrs(hight, width, CV_64F, cv::Scalar(0)); // 假设深度图是64位浮点数  
	cv::Mat brightptrs(hight, width, CV_8UC1, cv::Scalar(0));
	switch (hdr_num)
	{
	case 1:
	{
		depth = hdr_depth_list[0];
	}
	break;
	case 2:
	{
		

		cuda_merge_hdr_2_cpu(hdr_depth_list[id[0]], hdr_depth_list[id[1]],
			hdr_brightness_list[id[0]], hdr_brightness_list[id[1]],
			hight, width, depthptrs, brightptrs);
		

	}
	break;
	case 3:
	{
		cuda_merge_hdr_3_cpu(hdr_depth_list[id[0]], hdr_depth_list[id[1]], hdr_depth_list[id[2]],
			hdr_brightness_list[id[0]], hdr_brightness_list[id[1]], hdr_brightness_list[id[2]],
			hight, width, depthptrs, brightptrs);


	}
	break;
	case 4:
	{
		cuda_merge_hdr_4_cpu(hdr_depth_list[id[0]], hdr_depth_list[id[1]], hdr_depth_list[id[2]], hdr_depth_list[id[3]],
			hdr_brightness_list[id[0]], hdr_brightness_list[id[1]], hdr_brightness_list[id[2]], hdr_brightness_list[id[3]],
			hight, width, depthptrs, brightptrs);


	}
	break;
	case 5:
	{
		cuda_merge_hdr_5_cpu(hdr_depth_list[id[0]], hdr_depth_list[id[1]], hdr_depth_list[id[2]], hdr_depth_list[id[3]], hdr_depth_list[id[4]],
			hdr_brightness_list[id[0]], hdr_brightness_list[id[1]], hdr_brightness_list[id[2]], hdr_brightness_list[id[3]], hdr_brightness_list[id[4]],
			hight, width, depthptrs, brightptrs);


	}
	break;
	case 6:
	{
		cuda_merge_hdr_6_cpu(hdr_depth_list[id[0]], hdr_depth_list[id[1]], hdr_depth_list[id[2]], hdr_depth_list[id[3]], hdr_depth_list[id[4]], hdr_depth_list[id[5]],
			hdr_brightness_list[id[0]],hdr_brightness_list[id[1]], hdr_brightness_list[id[2]], hdr_brightness_list[id[3]], hdr_brightness_list[id[4]], hdr_brightness_list[id[5]],
			hight, width, depthptrs, brightptrs);


	}
	break;

	default:
		return false;

	}
	depth = depthptrs.clone();
	bright = brightptrs.clone();

	bright = hdr_brightness_list[hdr_num - 1];

	return true;
}




/********************************************************************************************************************************************/

bool cuda_copy_repetition_pattern_to_memory(unsigned char* patterns_ptr,int serial_flag)
{
	CHECK(hipMemcpyAsync(d_repetition_patterns_list_[serial_flag], patterns_ptr, h_image_height_*h_image_width_* sizeof(unsigned char), hipMemcpyHostToDevice));
}

bool cuda_merge_repetition_patterns(int repetition_serial)
{

	int merge_serial = repetition_serial%6; 
	kernel_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_patterns_list_[repetition_serial],h_image_height_, h_image_width_,d_repetition_merge_patterns_list_[merge_serial]);

	return true;
}


bool cuda_compute_merge_phase(int repetition_count)
{

	kernel_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_merge_patterns_list_[0], d_repetition_merge_patterns_list_[1],
		d_repetition_merge_patterns_list_[2],d_repetition_merge_patterns_list_[3],d_repetition_merge_patterns_list_[4],d_repetition_merge_patterns_list_[5] ,
		repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{

		kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,d_repetition_merge_patterns_list_[0], d_repetition_merge_patterns_list_[1],
		d_repetition_merge_patterns_list_[2],d_repetition_merge_patterns_list_[3],d_repetition_merge_patterns_list_[4],d_repetition_merge_patterns_list_[5] ,
		repetition_count, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
	}
 

	return true;
}


bool cuda_clear_repetition_02_patterns()
{
	for(int i = 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
				hipMemset(d_repetition_02_merge_patterns_list_[i], 0, h_image_height_ * h_image_width_ * sizeof(ushort));
				// CHECK(hipMemcpyAsync(d_repetition_02_merge_patterns_list_[i], &val,image_width_* image_height_*sizeof(ushort), hipMemcpyHostToDevice));
	}
	hipMemset(d_merge_brightness_map_, 0, h_image_height_ * h_image_width_ * sizeof(ushort));
	hipMemset(d_brightness_short_map_, 0, h_image_height_ * h_image_width_ * sizeof(ushort));
 
	// hipDeviceSynchronize();
  
  return true;
}

bool cuda_merge_repetition_02_patterns(int repetition_serial)
{
	if(0 == repetition_serial)
	{
		kernel_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_brightness_map_,
		h_image_height_, h_image_width_,d_merge_brightness_map_);
	}

 
	// int merge_serial = repetition_serial%19; 
	kernel_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_patterns_list_[repetition_serial],h_image_height_, h_image_width_,d_repetition_02_merge_patterns_list_[repetition_serial]);

	return true;
}
bool cuda_merge_repetition_02_patterns_cpu(int nr,int nc,int repetition_serial, std::vector<cv::Mat> patterns_, cv::Mat& merge_brightness_map,
	std::vector<cv::Mat>& repetition_02_merge_patterns_list)
{
	std::cout << "merge:" << std::endl;
	//cv::Mat brightness = merge_brightness_map.clone();
	//cv::Mat repetition = repetition_02_merge_patterns_list[repetition_serial].clone();
	//cv::Mat pattern = patterns_[repetition_serial];

	if (0 == repetition_serial)
	{
		for (int i = 0; i < nr; i++) {

			uchar* patterns0 = patterns_[repetition_serial].ptr<uchar>(i);
			ushort* merge_bright = merge_brightness_map.ptr<ushort>(i);
			for (int j = 0; j < nc; j++) {
				merge_bright[j] += patterns0[j];

			}
		}
	}

	for (int i = 0; i < nr; i++) {

		uchar* patterns = patterns_[repetition_serial].ptr<uchar>(i);
		ushort* repetition_02_merge_patterns = repetition_02_merge_patterns_list[repetition_serial].ptr<ushort>(i);
		for (int j = 0; j < nc; j++) {
			repetition_02_merge_patterns[j] += patterns[j];
		}
	}

	//merge_brightness_map = brightness.clone();
	//repetition_02_merge_patterns_list[repetition_serial] = repetition.clone();



	return true;
}


bool cuda_merge_repetition_02_patterns_16(unsigned short * const d_in_pattern,int repetition_serial)
{


	if (0 == repetition_serial)
	{
		
		CHECK(hipMemcpyAsync(d_brightness_short_map_, d_in_pattern, d_image_height_*d_image_width_* sizeof(unsigned short), hipMemcpyHostToDevice)); 

		kernel_merge_pattern_16<<<blocksPerGrid, threadsPerBlock>>>(d_brightness_short_map_,
																 h_image_height_, h_image_width_, d_merge_brightness_map_);
	}

	cv::Mat smooth_mat(d_image_height_, d_image_width_, CV_16UC1, d_in_pattern);
	if (7 < repetition_serial || 2 > repetition_serial)
	{
		LOG(INFO) << "Start GaussianBlur:";
		cv::GaussianBlur(smooth_mat, smooth_mat, cv::Size(5, 5), 1, 1);
		LOG(INFO) << "finished GaussianBlur!";
	}

	CHECK(hipMemcpyAsync(d_brightness_short_map_, smooth_mat.data, d_image_height_*d_image_width_* sizeof(unsigned short), hipMemcpyHostToDevice)); 
 
	// int merge_serial = repetition_serial%19;
	kernel_merge_pattern_16<<<blocksPerGrid, threadsPerBlock>>>(d_brightness_short_map_, h_image_height_, h_image_width_, 
	d_repetition_02_merge_patterns_list_[repetition_serial]);

	return true;
}

bool cuda_compute_merge_repetition_02_phase(int repetition_count,int phase_num)
{
	
	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],
		d_repetition_02_merge_patterns_list_[2],d_repetition_02_merge_patterns_list_[3],repetition_count, h_image_height_, h_image_width_,d_wrap_map_list_[0], d_confidence_map_list_[0]);
			
	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5],
		d_repetition_02_merge_patterns_list_[6],d_repetition_02_merge_patterns_list_[7],repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[1], d_confidence_map_list_[1]);

	kernel_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[8], d_repetition_02_merge_patterns_list_[9],
		d_repetition_02_merge_patterns_list_[10],d_repetition_02_merge_patterns_list_[11],repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[2], d_confidence_map_list_[2]);
	
	kernel_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[12], d_repetition_02_merge_patterns_list_[13],
		d_repetition_02_merge_patterns_list_[14],d_repetition_02_merge_patterns_list_[15],d_repetition_02_merge_patterns_list_[16],d_repetition_02_merge_patterns_list_[17] ,
		repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{

		kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_repetition_02_merge_patterns_list_[12], d_repetition_02_merge_patterns_list_[13],
		d_repetition_02_merge_patterns_list_[14],d_repetition_02_merge_patterns_list_[15],d_repetition_02_merge_patterns_list_[16],d_repetition_02_merge_patterns_list_[17],
																			   repetition_count, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
	}

	if(1 == phase_num)
	{
		kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_02_merge_patterns_list_[18],repetition_count,h_image_height_, h_image_width_,d_brightness_map_);
	}
	else if (2 == phase_num)
	{

		int i = 18;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count, h_image_height_, h_image_width_,d_wrap_map_list_[4], d_confidence_map_list_[4]);

		i = 22;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[5], d_confidence_map_list_[5]);

		i = 26;
		kernel_merge_four_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			   d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], repetition_count,h_image_height_, h_image_width_, d_wrap_map_list_[6], d_confidence_map_list_[6]);

		i = 30;
		kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			  d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], d_repetition_02_merge_patterns_list_[i + 4], d_repetition_02_merge_patterns_list_[i + 5],
																			  repetition_count, h_image_height_, h_image_width_, d_wrap_map_list_[7], d_confidence_map_list_[7]);

		if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		{

			kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_repetition_02_merge_patterns_list_[i + 0], d_repetition_02_merge_patterns_list_[i + 1],
																			  d_repetition_02_merge_patterns_list_[i + 2], d_repetition_02_merge_patterns_list_[i + 3], d_repetition_02_merge_patterns_list_[i + 4], d_repetition_02_merge_patterns_list_[i + 5],
																			  repetition_count,cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
		}

		kernel_merge_brigntness_map<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[36], repetition_count, h_image_height_, h_image_width_,d_brightness_map_);

		 
	}

	return true;
}

/********************************************************************************************************************************************/
//filter
void cuda_remove_points_base_radius_filter(float dot_spacing,float radius,int threshold_num)
{

	// cv::Mat pointcloud(1200, 1920, CV_32FC3, cv::Scalar(0));
	// CHECK(hipMemcpy(pointcloud.data, d_point_cloud_map_, 3 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// std::vector<cv::Mat> channels;
	// cv::split(pointcloud, channels);
	// cv::imwrite("depth_f.tiff", channels[2]);

	// hipDeviceSynchronize();
	LOG(INFO)<<"kernel_reconstruct_pointcloud_base_depth:"; 
	kernel_reconstruct_pointcloud_base_depth << <blocksPerGrid, threadsPerBlock >> > (h_image_width_,h_image_height_,d_undistort_map_x_,d_undistort_map_y_,
	d_camera_intrinsic_,d_camera_distortion_,d_depth_map_,d_point_cloud_map_);

	// hipDeviceSynchronize();

	// CHECK(hipMemcpy(pointcloud.data, d_point_cloud_map_, 3 * h_image_height_ * h_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// channels.clear();
	// cv::split(pointcloud, channels);
	// cv::imwrite("depth_e.tiff", channels[2]);
// hipDeviceSynchronize();
	// LOG(INFO) << "remove_base_radius_filter start:";

	// //相机像素为5.4um、焦距12mm。dot_spacing = 5.4*distance/12000 mm，典型值0.54mm（1200）

	float d2 = dot_spacing * dot_spacing;
	float r2 = radius * radius;

	// hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_filter_radius_outlier_removal), hipFuncCachePreferL1);

	// kernel_filter_radius_outlier_removal<<<blocksPerGrid, threadsPerBlock>>>(h_image_height_, h_image_width_, d_point_cloud_map_, d_mask_map_, d2, r2, threshold_num);
	// hipDeviceSynchronize();
	// LOG(INFO)<<"kernel_filter_radius_outlier_removal finished!";
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid((d_image_width_ - 1) / O_TILE_WIDTH + 1, (d_image_height_ - 1) / O_TILE_WIDTH + 1, 1);
    kernel_filter_radius_outlier_removal_shared << <dimGrid, dimBlock >> > (d_image_height_, d_image_width_, d_point_cloud_map_, d_mask_map_, d2, r2, threshold_num);
    // hipDeviceSynchronize();
	// LOG(INFO)<<"kernel_filter_radius_outlier_removal_shared finished!";

	// LOG(INFO) << "remove start:";
	kernel_removal_points_base_mask<<<blocksPerGrid, threadsPerBlock>>>(h_image_height_, h_image_width_, d_point_cloud_map_, d_depth_map_, d_mask_map_);

	// hipDeviceSynchronize();

	// LOG(INFO)<<"removal finished!";
}


void cuda_filter_reflect_noise(float * const unwrap_map)
{
    // dim3 threadsPerBlock_p(img_width);
    // dim3 blocksPerGrid_p(img_height);

	//按行来组织线程
    dim3 threadsPerBlock_p(4, 4);
    // dim3 blocksPerGrid_p(15,2);
    dim3 blocksPerGrid_p;
	if(1200 == h_image_height_)
	{
		blocksPerGrid_p.x = (40 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (30 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	else if(2048 == h_image_height_)
	{
		blocksPerGrid_p.x = (64 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (32 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}

 
 	kernel_filter_reflect_noise << <blocksPerGrid_p, threadsPerBlock_p >> > ( h_image_height_,h_image_width_, unwrap_map);
}


void fisher_filter(float fisher_confidence_val)
{
	//按行来组织线程
    dim3 threadsPerBlock_p(32, 1);
    dim3 blocksPerGrid_p;
	if(1200 == h_image_height_)//1920
	{
		blocksPerGrid_p.x = 1;
		blocksPerGrid_p.y = 1200;
	}
	else if(2048 == h_image_height_)//2448
	{
		blocksPerGrid_p.x = 1;
		blocksPerGrid_p.y = 2048;
	}
	hipDeviceSynchronize();
	LOG(INFO)<<"fisher start"; 
	kernel_fisher_filter <<< blocksPerGrid_p, threadsPerBlock_p >>> (h_image_height_, h_image_width_, (FISHER_CENTER_LOW + (fisher_confidence_val * FISHER_CENTER_RATE)), d_fisher_confidence_map, d_fisher_mask_, d_unwrap_map_list_[0]);
	hipDeviceSynchronize();
	LOG(INFO)<<"fisher end"; 
}

void phase_monotonicity_filter(float monotonicity_val)
{
	// 传入的monotonicity_val应当在（-10， 2）之间，-10 - monotonicity_val之间的被认为是噪声，传入参数是0 - 100之间的数字
	monotonicity_val = monotonicity_val / 100. - 0.5;
	//按照每个像素均独立的思想来组织线程
	hipDeviceSynchronize();
	LOG(INFO)<<"monotonicity start";
	kernel_monotonicity_filter <<< blocksPerGrid, threadsPerBlock >>> (h_image_height_, h_image_width_, -10, monotonicity_val, d_fisher_mask_, d_unwrap_map_list_[0]);
	hipDeviceSynchronize();
	kernel_removal_phase_base_mask <<< blocksPerGrid, threadsPerBlock >>> (h_image_height_, h_image_width_, d_unwrap_map_list_[0], d_fisher_mask_);
	hipDeviceSynchronize();
	LOG(INFO)<<"monotonicity end";
}

void depth_filter(float depth_threshold_val)
{
	dim3 threadsPerBlock_p(4, 4);
    dim3 blocksPerGrid_p;
	if(1200 == h_image_height_)
	{
		blocksPerGrid_p.x = (40 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (30 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	else if(2048 == h_image_height_)
	{
		blocksPerGrid_p.x = (64 + threadsPerBlock_p.x - 1) / threadsPerBlock_p.x;
		blocksPerGrid_p.y = (32 + threadsPerBlock_p.y - 1) / threadsPerBlock_p.y;
	}
	LOG(INFO)<<"depth filter start"; 
	kernel_depth_filter_step_1 <<< blocksPerGrid_p, threadsPerBlock_p >>> (h_image_height_, h_image_width_, depth_threshold_val, d_depth_map_, d_depth_map_temp_, d_fisher_mask_);//
	hipDeviceSynchronize();
	kernel_depth_filter_step_2 <<< blocksPerGrid_p, threadsPerBlock_p >>> (h_image_height_, h_image_width_, depth_threshold_val, d_depth_map_, d_depth_map_temp_, d_fisher_mask_);
	hipDeviceSynchronize();
	LOG(INFO)<<"depth filter end"; 
}

/****************************************************************************************************************************/
int cuda_copy_minsw8_pattern_to_memory_16(unsigned short* pattern_ptr,int serial_flag)
{
	if(serial_flag> 16)
	{
		return -1;
	}
 
	if(0 == serial_flag)
	{

		CHECK(hipMemcpyAsync(d_brightness_short_map_, pattern_ptr, d_image_height_*d_image_width_* sizeof(unsigned short), hipMemcpyHostToDevice)); 

		kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_brightness_short_map_,
							16,h_image_height_, h_image_width_,d_brightness_map_);
	}


	cv::Mat smooth_mat(d_image_height_, d_image_width_, CV_16UC1, pattern_ptr);
	if (7< serial_flag || serial_flag < 2)
	{
		LOG(INFO) << "Start GaussianBlur:";
		cv::GaussianBlur(smooth_mat, smooth_mat, cv::Size(5, 5), 1, 1);

		LOG(INFO) << "finished GaussianBlur!";
	}


	LOG(INFO) << "start copy:";
	// CHECK(hipMemcpyAsync(d_repetition_02_merge_patterns_list_[serial_flag], pattern_ptr, d_image_height_ * d_image_width_ * sizeof(unsigned short), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_repetition_02_merge_patterns_list_[serial_flag], smooth_mat.data, 
	d_image_height_ * d_image_width_ * sizeof(unsigned short), hipMemcpyHostToDevice));
	LOG(INFO) << "copy finished!";
}


/****************************************************************************************************************************/
int cuda_copy_minsw8_pattern_to_memory(unsigned char* pattern_ptr,int serial_flag)
{
	if(serial_flag> 16)
	{
		return -1;
	}

	cv::Mat smooth_mat(d_image_height_, d_image_width_, CV_8UC1, pattern_ptr);
	if (7 < serial_flag || 2> serial_flag)
	{
		LOG(INFO) << "Start GaussianBlur:";
		cv::GaussianBlur(smooth_mat, smooth_mat, cv::Size(5, 5), 1, 1); 
		LOG(INFO) << "finished GaussianBlur!";
	}
	LOG(INFO) << "start copy:";
	CHECK(hipMemcpyAsync(d_patterns_list_[serial_flag], smooth_mat.data, d_image_height_ * d_image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
	LOG(INFO) << "copy finished!";
}


int cuda_handle_model06_16()
{
	
	kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_02_merge_patterns_list_[0],16,
	h_image_height_, h_image_width_,d_brightness_map_);
	 

    kernel_generate_merge_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,
	d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],
	d_repetition_02_merge_patterns_list_[ThresholdMapSeries]);

	// cv::Mat threshold_map(d_image_height_,d_image_width_,CV_16F,cv::Scalar(0));
	// CHECK(hipMemcpy(threshold_map.data, d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
	//  1 * d_image_height_ * d_image_width_ * sizeof(ushort), hipMemcpyDeviceToHost));
	// cv::imwrite("threshold_map.tiff", threshold_map);
 
	// 六步相移
	kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																			  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																			  16, h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{

		kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																			  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																			  16, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
	}

	for(int i= 8;i<16;i++)
	{
		kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
																			d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
																			i - 8, d_patterns_list_[Minsw8MapSeries]);

		// if (0 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		// {
		// 	kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																		d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																		i - 8, d_patterns_list_[Minsw8MapSeries]);
		// }
		// else
		// {

		// 	kernel_threshold_merge_patterns_with_uncertain<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																					   d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																					   i - 8, d_patterns_list_[Minsw8MapSeries], d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
		// }
	}

	kernel_minsw8_to_bin<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
	 d_minsw8_table_, d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

	kernel_bin_unwrap<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[binMapSeries], 
	d_wrap_map_list_[3], d_unwrap_map_list_[0]);

	return DF_SUCCESS;
}


int cuda_handle_repetition_model06_16(int repetition_count)
{
	 
	kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_merge_brightness_map_,
	repetition_count*16,h_image_height_, h_image_width_,d_brightness_map_);

	// cv::Mat brigntness_map(d_image_height_,d_image_width_,CV_8UC1,cv::Scalar(0));
	// CHECK(hipMemcpy(brigntness_map.data, d_brightness_map_, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
	// cv::imwrite("brigntness_map.bmp", brigntness_map);

    kernel_generate_merge_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,
	d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],d_repetition_02_merge_patterns_list_[ThresholdMapSeries]);
 
	// 六步相移
	kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																			  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																			  repetition_count, h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	// cv::Mat wrap_map(d_image_height_,d_image_width_,CV_32FC1,cv::Scalar(0));
	// CHECK(hipMemcpy(wrap_map.data, d_wrap_map_list_[3], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	// cv::imwrite("wrap_map.tiff", wrap_map);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{

		kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																			   d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																			   repetition_count, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_,d_uncertain_map_);
	}

	//相位校正
				//相位校正
	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_gray_rectify)
	{
		cv::Mat convolution_kernal = cv::getGaussianKernel(cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r,
														   cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_sigma * 0.02, CV_32F);
		convolution_kernal = convolution_kernal * convolution_kernal.t();
		cuda_copy_convolution_kernal_to_memory((float *)convolution_kernal.data,
											   cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
		cuda_rectify_six_step_pattern_phase(3, cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
	}
 
	for(int i= 8;i<16;i++)
	{
		kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
																			d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
																			i - 8, d_patterns_list_[Minsw8MapSeries]);

		// if (0 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		// {
		// 	kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																		d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																		i - 8, d_patterns_list_[Minsw8MapSeries]);
		// }
		// else
		// {

		// 	kernel_threshold_merge_patterns_with_uncertain<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																					   d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																					   i - 8, d_patterns_list_[Minsw8MapSeries], d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
		// }
	}

	kernel_minsw8_to_bin<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_minsw8_table_, d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

	kernel_bin_unwrap<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[binMapSeries], d_wrap_map_list_[3], d_unwrap_map_list_[0]);


// hipDeviceSynchronize();
	return DF_SUCCESS;
}


int cuda_handle_repetition_model06(int repetition_count)
{
 
	kernel_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_merge_brightness_map_,
	repetition_count,h_image_height_, h_image_width_,d_brightness_map_);
	 

    kernel_generate_merge_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,
	d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],d_repetition_02_merge_patterns_list_[ThresholdMapSeries]);

	// 六步相移
	kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																		  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																		  repetition_count, h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
	{

		kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																		  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																		repetition_count, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_,d_uncertain_map_);
	}

	//相位校正
				//相位校正
	if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_gray_rectify)
	{
		cv::Mat convolution_kernal = cv::getGaussianKernel(cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r,
														   cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_sigma * 0.02, CV_32F);
		convolution_kernal = convolution_kernal * convolution_kernal.t();
		cuda_copy_convolution_kernal_to_memory((float *)convolution_kernal.data,
											   cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
		cuda_rectify_six_step_pattern_phase(2, cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
	}

	for (int i = 8; i < 16; i++)
	{
		kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
																			d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
																			i - 8, d_patterns_list_[Minsw8MapSeries]);

		// if (0 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		// {
		// 	kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																		d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																		i - 8, d_patterns_list_[Minsw8MapSeries]);
		// }
		// else
		// {

		// 	kernel_threshold_merge_patterns_with_uncertain<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																		d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																		i - 8, d_patterns_list_[Minsw8MapSeries], d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
		// }
	}

	kernel_minsw8_to_bin<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_minsw8_table_, d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

	kernel_bin_unwrap<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[binMapSeries], d_wrap_map_list_[3], d_unwrap_map_list_[0]);

	return 0;
}


int cuda_handle_repetition_model06_cpu(int repetition_count, int nr, int nc,  std::vector<cv::Mat> repetition_02_merge_patterns_list,
	cv::Mat& threshold_map, cv::Mat& mask, cv::Mat& wrap, cv::Mat& sw_k_map, cv::Mat& minsw_map, cv::Mat& k2_map, cv::Mat& unwrap)
{
	DF_Encode encode;
	cv::Mat map_white_map = repetition_02_merge_patterns_list[0].clone();
	cv::Mat map_black_map = repetition_02_merge_patterns_list[1].clone();


	for (int r = 0; r < nr; r++)
	{
		ushort* ptr_b = map_black_map.ptr<ushort>(r);
		ushort* ptr_w = map_white_map.ptr<ushort>(r);
		ushort* ptr_t = threshold_map.ptr<ushort>(r);
		//uchar* ptr_c = threshold_confidence.ptr<uchar>(r);
		for (int c = 0; c < nc; c++)
		{
			ushort d = ptr_w[c] - ptr_b[c];
			ptr_t[c] = ptr_b[c] + 0.5 + d / 2.0;
			//ptr_c[c] = std::abs(d);
		}
	}


	//六步相移数据6张
	std::vector<cv::Mat> phase_shift_patterns_img(repetition_02_merge_patterns_list.begin() + 2, repetition_02_merge_patterns_list.begin() + 8);
	bool ret = encode.computePhaseShift_repetition(phase_shift_patterns_img, wrap, mask);



	std::vector<cv::Mat> minsw_gray_code_patterns_img(repetition_02_merge_patterns_list.begin() + 8, repetition_02_merge_patterns_list.begin() + 8 + 8);
	bool ret1 = encode.decodeMinswGrayCode_repetition(minsw_gray_code_patterns_img,threshold_map, sw_k_map);

	for (int r = 0; r < nr; r++)
	{
		float* ptr_sw = minsw_map.ptr<float>(r);
		uchar* ptr_k2 = sw_k_map.ptr<uchar>(r);
		for (int c = 0; c < nc; c++)
		{
			int bin_value = -1;
			bool ret = encode.minsw8CodeToValue(ptr_k2[c], bin_value);
			ptr_sw[c] = bin_value;
		}
	}

	minsw_map.convertTo(k2_map, CV_8U);
	encode.unwrapBase2Kmap_repetition(wrap, k2_map, unwrap);


	return 0;
}



/**********************************************************************************************************************/


int cuda_handle_minsw8_16(int flag)
{
	
    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid((d_image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
    (d_image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);

	switch(flag)
	{
		case 2:
		{ 
            //生成阈值图
            //  kernel_generate_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[0], d_patterns_list_[1],d_patterns_list_[ThresholdMapSeries]);
			kernel_generate_merge_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,
			d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],
			d_repetition_02_merge_patterns_list_[ThresholdMapSeries]);
			 
        } 
    		break;
        case 8:
		{ 

				//六步相移
				// int i= 2; 
				// kernel_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0],
				// d_patterns_list_[i + 1], d_patterns_list_[i + 2],d_patterns_list_[i + 3],d_patterns_list_[i + 4],d_patterns_list_[i + 5]
				// ,d_wrap_map_list_[3], d_confidence_map_list_[3]);

				kernel_merge_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																					  d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																					  16, h_image_height_, h_image_width_, d_wrap_map_list_[3], d_confidence_map_list_[3]);

				if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
				{

					kernel_merge_computre_global_light<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_repetition_02_merge_patterns_list_[2], d_repetition_02_merge_patterns_list_[3],
																						   d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5], d_repetition_02_merge_patterns_list_[6], d_repetition_02_merge_patterns_list_[7],
																						   16, cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b, d_direct_light_map_, d_global_light_map_,d_uncertain_map_);
				}

			//相位校正
			if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_gray_rectify)
            {
                cv::Mat convolution_kernal = cv::getGaussianKernel(cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r, 
				cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_sigma * 0.02, CV_32F);
	            convolution_kernal = convolution_kernal * convolution_kernal.t();
                cuda_copy_convolution_kernal_to_memory((float*)convolution_kernal.data, 
				cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
                cuda_rectify_six_step_pattern_phase(3, cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
 
				/*********************************************************************************************************/
            }
 
 
        } 
    		break;
 
  
		default :
			break;
	}

	if (flag > 7 && flag < 16)
	{
		// kernel_threshold_patterns << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[flag], d_patterns_list_[ThresholdMapSeries],
		// flag-8,d_patterns_list_[Minsw8MapSeries]);

		// if (0 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		{
			kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
																				d_repetition_02_merge_patterns_list_[flag], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
																				flag - 8, d_patterns_list_[Minsw8MapSeries]);

			// kernel_threshold_merge_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
			// 																	d_repetition_02_merge_patterns_list_[i], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
			// 																	i - 8, d_patterns_list_[Minsw8MapSeries]);
		}
		// else
		// {

		// 	kernel_threshold_merge_patterns_with_uncertain<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		// 																					   d_repetition_02_merge_patterns_list_[flag], d_repetition_02_merge_patterns_list_[ThresholdMapSeries],
		// 																					   flag - 8, d_patterns_list_[Minsw8MapSeries], d_direct_light_map_, d_global_light_map_, d_uncertain_map_);
		// }
	}

	if(15 == flag)
    {
        // kernel_minsw8_to_bin << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_minsw8_table_,d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

        // kernel_bin_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_patterns_list_[binMapSeries],d_wrap_map_list_[3],d_unwrap_map_list_[0]);
		
		kernel_minsw8_to_bin<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_,
		d_minsw8_table_, d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

		kernel_bin_unwrap<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[binMapSeries], 
		d_wrap_map_list_[3], d_unwrap_map_list_[0]);


		// hipDeviceSynchronize();
		// cv::Mat phase(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
		// CHECK(hipMemcpy(phase.data, d_patterns_list_[binMapSeries], 1 * d_image_height_ * d_image_width_ * sizeof(uchar), hipMemcpyDeviceToHost));
		// cv::imwrite("code.bmp", phase);

		// cv::Mat threshold_map(d_image_height_, d_image_width_, CV_32F, cv::Scalar(0));
		// CHECK(hipMemcpy(threshold_map.data, d_unwrap_map_list_[0], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
		// cv::imwrite("threshold_map.tiff", threshold_map);
	}

    return 0;
}




/*************************************************************************************************************************/

 int cuda_handle_minsw8(int flag)
 {

    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid((d_image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
    (d_image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);

	switch(flag)
	{
		case 2:
		{ 
            //生成阈值图
             kernel_generate_threshold_map << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[0], d_patterns_list_[1],d_patterns_list_[ThresholdMapSeries]);
 
			 
        } 
    		break;
        case 8:
		{ 

				//六步相移
				int i= 2;

				// kernel_six_step_phase_shift_with_average<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[i + 0],
				// 																			 d_patterns_list_[i + 1], d_patterns_list_[i + 2], d_patterns_list_[i + 3], d_patterns_list_[i + 4],
				// 																			 d_patterns_list_[i + 5], d_wrap_map_list_[3], d_confidence_map_list_[3], d_patterns_list_[ThresholdMapSeries], d_brightness_map_);

				kernel_six_step_phase_shift<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[i + 0],
																				d_patterns_list_[i + 1], d_patterns_list_[i + 2], d_patterns_list_[i + 3], d_patterns_list_[i + 4], d_patterns_list_[i + 5], d_wrap_map_list_[3], d_confidence_map_list_[3]);

				if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
				{
					// kernel_six_step_phase_shift_global << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_patterns_list_[i+0],
					// d_patterns_list_[i + 1], d_patterns_list_[i + 2],d_patterns_list_[i + 3],d_patterns_list_[i + 4],d_patterns_list_[i + 5]
					// ,d_wrap_map_list_[3], d_confidence_map_list_[3],0.25,
					// d_direct_light_map_,d_global_light_map_); //cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b

					kernel_computre_global_light_with_background<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[i + 0],
																					 d_patterns_list_[i + 1], d_patterns_list_[i + 2], d_patterns_list_[i + 3], d_patterns_list_[i + 4], d_patterns_list_[i + 5],
																					d_patterns_list_[0],d_patterns_list_[1] ,cuda_system_config_settings_machine_.Instance().firwmare_param_.global_light_filter_b,
																					  d_direct_light_map_, d_global_light_map_,d_uncertain_map_);

					// hipDeviceSynchronize();
					// cv::Mat black_map(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
					// CHECK(hipMemcpy(black_map.data, d_patterns_list_[1], 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
					// cv::imwrite("black_map.bmp", black_map);

					// cv::Mat direct_map(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
					// CHECK(hipMemcpy(direct_map.data, d_direct_light_map_, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
					// cv::imwrite("direct_map.bmp", direct_map);

					// cv::Mat global_map(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
					// CHECK(hipMemcpy(global_map.data, d_global_light_map_, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
					// cv::imwrite("global_map.bmp", global_map);
				}

			//相位校正
			if (1 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_gray_rectify)
            {
                cv::Mat convolution_kernal = cv::getGaussianKernel(cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r, 
				cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_sigma * 0.02, CV_32F);
	            convolution_kernal = convolution_kernal * convolution_kernal.t();
                cuda_copy_convolution_kernal_to_memory((float*)convolution_kernal.data, 
				cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
                cuda_rectify_six_step_pattern_phase(2, cuda_system_config_settings_machine_.Instance().firwmare_param_.gray_rectify_r);
 
				/*********************************************************************************************************/
            }
 
 
        } 
    		break;
 
  
		default :
			break;
	}


    if(flag> 7 && flag< 16)
    {

		// if (0 == cuda_system_config_settings_machine_.Instance().firwmare_param_.use_global_light_filter)
		{
			kernel_threshold_patterns<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[flag], d_patterns_list_[ThresholdMapSeries],
																		  flag - 8, d_patterns_list_[Minsw8MapSeries]);
		}
		// else
		// {

		// 	kernel_threshold_patterns_with_uncertain<<<blocksPerGrid, threadsPerBlock>>>(d_image_width_, d_image_height_, d_patterns_list_[flag], d_patterns_list_[ThresholdMapSeries],
		// 																				 flag - 8, d_patterns_list_[Minsw8MapSeries], d_direct_light_map_, d_global_light_map_, d_uncertain_map_);

			// hipDeviceSynchronize();

			// cv::Mat uncertain_map(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
			// CHECK(hipMemcpy(uncertain_map.data, d_uncertain_map_, 1 * d_image_height_ * d_image_width_ * sizeof(char), hipMemcpyDeviceToHost));
			// cv::imwrite("uncertain_map.bmp", uncertain_map);
		// }
	}

    if(15 == flag)
    {
        kernel_minsw8_to_bin << <blocksPerGrid, threadsPerBlock >> > (d_image_width_,d_image_height_,d_minsw8_table_,d_patterns_list_[Minsw8MapSeries], d_patterns_list_[binMapSeries]);

        kernel_bin_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_image_width_,d_image_height_,d_patterns_list_[binMapSeries],d_wrap_map_list_[3],d_unwrap_map_list_[0]);

		// hipDeviceSynchronize();
		// cv::Mat phase(d_image_height_, d_image_width_, CV_8U, cv::Scalar(0));
		// CHECK(hipMemcpy(phase.data, d_patterns_list_[binMapSeries], 1 * d_image_height_ * d_image_width_ * sizeof(uchar), hipMemcpyDeviceToHost));
		// cv::imwrite("code.bmp", phase);

		// cv::Mat threshold_map(d_image_height_, d_image_width_, CV_32F, cv::Scalar(0));
		// CHECK(hipMemcpy(threshold_map.data, d_unwrap_map_list_[0], 1 * d_image_height_ * d_image_width_ * sizeof(float), hipMemcpyDeviceToHost));
		// cv::imwrite("threshold_map.tiff", threshold_map);
	}

    return 0;
 }


 int cuda_handle_minsw8_cpu(int nr,int nc,int flag, std::vector<cv::Mat>& patterns_,cv::Mat& threshold_map, cv::Mat& mask, cv::Mat& wrap, cv::Mat& sw_k_map, cv::Mat& minsw_map, cv::Mat& k2_map, cv::Mat& unwrap)
 {

	 DF_Encode encode;

	 switch (flag)
	 {
	 case 2:
	 {
		 cv::Mat map_white_map = patterns_[0].clone();
		 cv::Mat map_black_map = patterns_[1].clone();
		 int nr = patterns_[0].rows;
		 int nc = patterns_[0].cols;

		 for (int r = 0; r < nr; r++)
		 {
			 uchar* ptr_b = map_black_map.ptr<uchar>(r);
			 uchar* ptr_w = map_white_map.ptr<uchar>(r);
			 uchar* ptr_t = threshold_map.ptr<uchar>(r);
			 //uchar* ptr_c = threshold_confidence.ptr<uchar>(r);
			 for (int c = 0; c < nc; c++)
			 {
				 float d = ptr_w[c] - ptr_b[c];
				 ptr_t[c] = ptr_b[c] + 0.5 + d / 2.0;
				 //ptr_c[c] = std::abs(d);
			 }
		 }

	 }
	 break;
	 case 8:
	 {


		 //六步相移数据6张
		 std::vector<cv::Mat> phase_shift_patterns_img(patterns_.begin() + 2, patterns_.begin() + 8);
		 bool ret = encode.computePhaseShift(phase_shift_patterns_img, wrap, mask);


	 }
	 break;


	 default:
		 break;
	 }


	 if (flag > 7 && flag < 16)
	 {

		 //第三步GPU中minsw生成Minsw8MapSeries17
	
		 //minsw数据8张
		 int space = flag - 8;
		bool ret1 = encode.decodeMinswGrayCode(patterns_[flag],space, threshold_map, sw_k_map);
	 }

	 if (15 == flag)
	 {

		 for (int r = 0; r < nr; r++)
		 {
			 float* ptr_sw = minsw_map.ptr<float>(r);
			 ushort* ptr_k2 = sw_k_map.ptr<ushort>(r);
			 for (int c = 0; c < nc; c++)
			 {
				 int bin_value = -1;
				 bool ret = encode.minsw8CodeToValue(ptr_k2[c], bin_value);
				 ptr_sw[c] = bin_value;
			 }
		 }

		 minsw_map.convertTo(k2_map, CV_16U);

		 encode.unwrapBase2Kmap(wrap, k2_map, unwrap);


	 }

	 return 0;
 }



/*****************************************************************************************************************************************************/
//repetition

void cuda_copy_phase_from_cuda_memory(float* phase_x,float* phase_y)
{
	CHECK(hipMemcpy(phase_x, d_unwrap_map_list_[0], d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
	CHECK(hipMemcpy(phase_y, d_unwrap_map_list_[1], d_image_height_*d_image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
}




/*****************************************************************************************************************************************************/









