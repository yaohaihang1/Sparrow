#include "hip/hip_runtime.h"
#include "reconstruct.cuh"
#include <opencv2/core.hpp> 
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include<iostream>
__device__ float d_confidence_ = 10;

__device__ int d_dlp_width_ = 0;
__device__ int d_dlp_height_ = 0;
__device__ float d_max_phase_ = 2* CV_PI; 

__device__ float d_min_z_ = 10; 
__device__ float d_max_z_ = 4500;


 #define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}
 
 
bool cuda_set_param_dlp_resolution(int width,int height)
{
	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &width, sizeof(float));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &height, sizeof(float));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	return true;
}
 
bool cuda_set_param_z_range(float min,float max)
{
	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_min_z_), &min, sizeof(float));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_max_z_), &max, sizeof(float));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	return true;
}


bool cuda_set_param_confidence(float val)
{
	hipError_t error_code = hipMemcpyToSymbol(HIP_SYMBOL(d_confidence_), &val, sizeof(float));

	if(error_code!= hipSuccess)
	{
		return false;
	}

	return true;
}

__device__ float bilinear_interpolation(float x, float y, int map_width, float *mapping)
{

	int x1 = floor(x);
	int y1 = floor(y);
	int x2 = x1 + 1;
	int y2 = y1 + 1;

	//row-y,col-x

	if (x1 == 1919) {
		float out = mapping[y1 *map_width + x1];
		return out;
	}
	else {
		float fq11 = mapping[y1 *map_width + x1];
		float fq21 = mapping[y1 *map_width + x2];
		float fq12 = mapping[y2 *map_width + x1];
		float fq22 = mapping[y2 *map_width + x2];

		if (-2 == fq11 || -2 == fq21 || -2 == fq12 || -2 == fq22)
		{
			return -2;
		}

		float out = fq11 * (x2 - x) * (y2 - y) + fq21 * (x - x1) * (y2 - y) + fq12 * (x2 - x) * (y - y1) + fq22 * (x - x1) * (y - y1);

		return out;
	}
	 

}

__global__ void kernel_remove_mask_result(int width, int height, unsigned char *const mask, uchar threshold, float *const depth, float *const pointcloud)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		/****************************************************************************/

		if (mask[offset] > threshold)
		{
			pointcloud[3 * offset + 0] = 0;
			pointcloud[3 * offset + 1] = 0;
			pointcloud[3 * offset + 2] = 0;
			depth[offset] = 0;
		}

		/******************************************************************/
	}
}

__global__ void kernel_reconstruct_pointcloud_base_table(int width,int height,float * const xL_rotate_x,float * const xL_rotate_y,float * const single_pattern_mapping,float * const R_1,float b,
float * const confidence_map,float * const phase_x , float * const pointcloud,float * const depth)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

  
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		/****************************************************************************/
		//phase to position
		float Xp = phase_x[offset] * d_dlp_width_ /d_max_phase_; 
		// float Xp = (phase_x[offset] * 1280) /(2*CV_PI); 
        float Xcr = xL_rotate_x[offset];
        float Ycr = xL_rotate_y[offset];
 
 
        float Xpr = bilinear_interpolation(Xp, (Ycr + 1) * 2000, 2000, single_pattern_mapping);
        float delta_X = std::abs(Xcr - Xpr); 
        float Z = b / delta_X;
	
		float X_L = Z * Xcr * R_1[0] + Z * Ycr * R_1[1] + Z * R_1[2];
		float Y_L = Z * Xcr * R_1[3] + Z * Ycr * R_1[4] + Z * R_1[5];
		float Z_L = Z * Xcr * R_1[6] + Z * Ycr * R_1[7] + Z * R_1[8];
 
  
		if(confidence_map[offset] > d_confidence_ && Z_L > d_min_z_ && Z_L< d_max_z_ && Xp > 0)
		// if(confidence_map[offset] > 10 && Z_L > 10 && Z_L< 3000 && Xp > 0)
		{
		    pointcloud[3 * offset + 0] = X_L;
		    pointcloud[3 * offset + 1] = Y_L;
		    pointcloud[3 * offset + 2] = Z_L; 
			
		    depth[offset] = Z_L; 
		}
		else
		{
		    pointcloud[3 * offset + 0] = 0;
		    pointcloud[3 * offset + 1] = 0;
		    pointcloud[3 * offset + 2] = 0; 
			
		    depth[offset] = 0; 
		}

		
		if (-2 == Xcr || -2 == Ycr || -2 == Xpr)
		{
			pointcloud[3 * offset + 0] = 0;
		    pointcloud[3 * offset + 1] = 0;
		    pointcloud[3 * offset + 2] = 0; 
			
		    depth[offset] = 0; 
		}
  
		/******************************************************************/


	}
}

__device__ float mini_bilinear_interpolation(float x, float y, int map_width, float *mapping)
{
	//map_width = 129;

	//先找到这个点所对应的mini中的四个角点
	//然后将这四个点算出来
	//最后双线性插值

	int index_x1 = floor(x / 16);
	int index_y1 = floor((y-1301) / 16);
	int index_x2 = index_x1 + 1;
	int index_y2 = index_y1 + 1;

	int x1 = index_x1 * 16;
	int y1 = index_y1 * 16 + 1301;
	int x2 = x1 + 16;
	int y2 = y1 + 16;

	//因为我生成的表比原来大，所以无需考虑边界条件
	//fq_xy
	float fq11 = mapping[index_y1 *map_width + index_x1];
	float fq21 = mapping[index_y1 *map_width + index_x2];
	float fq12 = mapping[index_y2 *map_width + index_x1];
	float fq22 = mapping[index_y2 *map_width + index_x2];

	float out = (fq11 * (x2 - x) * (y2 - y) + fq21 * (x - x1) * (y2 - y) + fq12 * (x2 - x) * (y - y1) + fq22 * (x - x1) * (y - y1))/256.;

	return out;
}


__global__ void kernel_reconstruct_pointcloud_base_minitable(uint32_t img_height, uint32_t img_width, float* const xL_rotate_x, float* const xL_rotate_y, float* const single_pattern_minimapping, float* const R_1, float b, 
 float* const confidence_map, float* const phase_x,float* const pointcloud, float* const depth)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;


	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		/****************************************************************************/
		//phase to position
		// float Xp = phase_x[serial_id] * 1280.0 / (128.0 * 2 * DF_PI);
		float Xp = phase_x[serial_id] * d_dlp_width_ /d_max_phase_; 

        float Xcr = xL_rotate_x[serial_id];
        float Ycr = xL_rotate_y[serial_id];
		// float Xcr = bilinear_interpolation(idx, idy, 1920, xL_rotate_x);
		// float Ycr = bilinear_interpolation(idx, idy, 1920, xL_rotate_y);
		//修改此处即可，需要自己写一个函数去查表
		float Xpr = mini_bilinear_interpolation(Xp, (Ycr + 1) * 2000, 128, single_pattern_minimapping);
		float delta_X = std::abs(Xcr - Xpr);
		float Z = b / delta_X;

		float X_L = Z * Xcr * R_1[0] + Z * Ycr * R_1[1] + Z * R_1[2];
		float Y_L = Z * Xcr * R_1[3] + Z * Ycr * R_1[4] + Z * R_1[5];
		float Z_L = Z * Xcr * R_1[6] + Z * Ycr * R_1[7] + Z * R_1[8];


		if(Z_L > d_min_z_ && Z_L< d_max_z_ && Xp > 0)
		// if (confidence_map[serial_id] > 10 && Z_L > 100 && Z_L < 2000)
		{
			pointcloud[3 * serial_id + 0] = X_L;
			pointcloud[3 * serial_id + 1] = Y_L;
			pointcloud[3 * serial_id + 2] = Z_L;

			depth[serial_id] = Z_L;
		}
		else
		{
			pointcloud[3 * serial_id + 0] = 0;
			pointcloud[3 * serial_id + 1] = 0;
			pointcloud[3 * serial_id + 2] = 0;

			depth[serial_id] = 0;
		}

		/******************************************************************/


	}
}


__device__ void undistortPoint(float x, float y,
	float fc_x, float fc_y,
	float cc_x, float cc_y,
	float k1, float k2, float k3, float p1, float p2,
	float& x_undistort, float& y_undistort)
{
	float x_distort = (x - cc_x) / fc_x;
	float y_distort = (y - cc_y) / fc_y;

	float x_iter = x_distort;
	float y_iter = y_distort;

	for (int i = 0; i < 20; i++)
	{
		float r_2 = x_iter * x_iter + y_iter * y_iter;
		float r_4 = r_2 * r_2;
		float r_6 = r_4 * r_2;
		float k_radial = 1 + k1 * r_2 + k2 * r_4 + k3 * r_6;
		float delta_x = 2 * p1 * x_iter * y_iter + p2 * (r_2 + 2 * x_iter * x_iter);
		float delta_y = p1 * (r_2 + 2 * y_iter * y_iter) + 2 * p2 * x_iter * y_iter;
		x_iter = (x_distort - delta_x) / k_radial;
		y_iter = (y_distort - delta_y) / k_radial;
	}
	x_undistort = x_iter * fc_x + cc_x;
	y_undistort = y_iter * fc_y + cc_y;

	//x_undistort = x_iter;
	//y_undistort = y_iter;

	return;
}


__global__ void kernel_reconstruct_pointcloud_base_depth(int width,int height,float * const xL_rotate_x,float * const xL_rotate_y,
float* const camera_intrinsic,float* const camera_distortion, float * const depth, float * const pointcloud)
 {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;


	const unsigned int serial_id = idy * width + idx;

	float camera_fx = camera_intrinsic[0];
	float camera_fy = camera_intrinsic[4];

	float camera_cx = camera_intrinsic[2];
	float camera_cy = camera_intrinsic[5]; 

	// float k1 = camera_distortion[0];
	// float k2 = camera_distortion[1];
	// float p1 = camera_distortion[2];
	// float p2 = camera_distortion[3];
	// float k3 = camera_distortion[4];

	// printf("camera_fx: %f",camera_fx);
	// printf("camera_fy: %f",camera_fy);
	// printf("k1: %f",k1);
	// printf("k2: %f",k2);
	// printf("k3: %f",k3);

	if (idx < width && idy < height)
	{

		if (depth[serial_id] > 0)
		{
			float undistort_x = xL_rotate_x[serial_id];
			float undistort_y = xL_rotate_y[serial_id];
			// float undistort_x = idx;
			// float undistort_y = idy;
			// undistortPoint(idx, idy, camera_intrinsic[0], camera_intrinsic[4], camera_intrinsic[2], camera_intrinsic[5],
			// camera_distortion[0], camera_distortion[1], camera_distortion[4], camera_distortion[2], camera_distortion[3],
			// undistort_x, undistort_y);

			pointcloud[3 * serial_id + 0] = (undistort_x - camera_cx) * depth[serial_id] / camera_fx;
			pointcloud[3 * serial_id + 1] = (undistort_y - camera_cy) * depth[serial_id] / camera_fy;
			pointcloud[3 * serial_id + 2] = depth[serial_id];
		}
		else
		{
			pointcloud[3 * serial_id + 0] = 0;
			pointcloud[3 * serial_id + 1] = 0;
			pointcloud[3 * serial_id + 2] = 0;
		}
	}
 }